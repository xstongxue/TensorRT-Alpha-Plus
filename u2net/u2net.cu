#include "hip/hip_runtime.h"
#include"u2net.h"
#include <thrust/extrema.h> // u2net
#include <thrust/device_ptr.h>

u2net::U2NET::U2NET(const utils::InitParameter& param) : m_param(param)
{
    // input
    m_input_src_device = nullptr;
    m_input_rgb_device = nullptr;
    m_input_resize_device = nullptr;
    m_input_norm_device = nullptr;
    m_input_hwc_device = nullptr;
    m_max_val_device = nullptr;
    m_min_val_device = nullptr;
    CHECK(hipMalloc(&m_input_src_device,    param.batch_size * 3 * param.src_h * param.src_w * sizeof(float)));
    CHECK(hipMalloc(&m_input_rgb_device,    param.batch_size * 3 * param.src_h * param.src_w * sizeof(float)));
    CHECK(hipMalloc(&m_input_resize_device, param.batch_size * 3 * param.dst_h * param.dst_h * sizeof(float)));
    CHECK(hipMalloc(&m_input_norm_device,   param.batch_size * 3 * param.dst_h * param.dst_h * sizeof(float)));
    CHECK(hipMalloc(&m_input_hwc_device,    param.batch_size * 3 * param.dst_h * param.dst_h * sizeof(float)));
    CHECK(hipMalloc(&m_max_val_device,      param.batch_size  * sizeof(float)));
    CHECK(hipMalloc(&m_min_val_device,      param.batch_size  * sizeof(float)));

    // output
    m_output_src_device = nullptr;
    m_output_resize_device = nullptr;
    m_output_resize_host = nullptr;
    m_output_mask_host = nullptr;

    CHECK(hipMalloc(&m_output_resize_device,    param.batch_size * 1 * param.src_h * param.src_w * sizeof(float)));
    m_output_resize_host = new float[param.batch_size * 1 * param.src_h * param.src_w];
    m_output_mask_host = new float[param.src_h * param.src_w];
}

u2net::U2NET::~U2NET()
{
    // input
    CHECK(hipFree(m_input_src_device));
    CHECK(hipFree(m_input_rgb_device));
    CHECK(hipFree(m_input_resize_device));
    CHECK(hipFree(m_input_norm_device));
    CHECK(hipFree(m_input_hwc_device));
    CHECK(hipFree(m_max_val_device));
    CHECK(hipFree(m_min_val_device));

    // output
    CHECK(hipFree(m_output_src_device));
    CHECK(hipFree(m_output_resize_device));
    delete[] m_output_resize_host;
    delete[] m_output_mask_host;
}

bool u2net::U2NET::init(const std::vector<unsigned char>& trtFile)
{
    if (trtFile.empty())
    {
        return false;
    }
    std::unique_ptr<nvinfer1::IRuntime> runtime =
        std::unique_ptr<nvinfer1::IRuntime>(nvinfer1::createInferRuntime(sample::gLogger.getTRTLogger()));
    if (runtime == nullptr)
    {
        return false;
    }
    this->m_engine = std::unique_ptr<nvinfer1::ICudaEngine>(runtime->deserializeCudaEngine(trtFile.data(), trtFile.size()));

    if (this->m_engine == nullptr)
    {
        return false;
    }
    this->m_context = std::unique_ptr<nvinfer1::IExecutionContext>(this->m_engine->createExecutionContext());
    if (this->m_context == nullptr)
    {
        return false;
    }
    this->m_context->setBindingDimensions(0, nvinfer1::Dims4(m_param.batch_size, 3, m_param.dst_h, m_param.dst_w));
    m_output_dims = this->m_context->getBindingDimensions(1);
    m_output_area = 1;
    for (int i = 1; i < m_output_dims.nbDims; i++)
    {
        if (m_output_dims.d[i] != 0)
        {
            m_output_area *= m_output_dims.d[i];
        }
    }
    CHECK(hipMalloc(&m_output_src_device, m_param.batch_size * m_output_area * sizeof(float)));
    float scale_y = float(m_param.dst_h) / m_param.src_h;
    float scale_x = float(m_param.dst_w) / m_param.src_w;
    cv::Mat src2dst = (cv::Mat_<float>(2, 3) << scale_x, 0.f, (-scale_x * m_param.src_w + m_param.dst_w + scale_x - 1) * 0.5,
        0.f, scale_y, (-scale_y * m_param.src_h + m_param.dst_h + scale_y - 1) * 0.5);
    cv::Mat dst2src = cv::Mat::zeros(2, 3, CV_32FC1);
    cv::invertAffineTransform(src2dst, dst2src);

    m_dst2src.v0 = dst2src.ptr<float>(0)[0];
    m_dst2src.v1 = dst2src.ptr<float>(0)[1];
    m_dst2src.v2 = dst2src.ptr<float>(0)[2];
    m_dst2src.v3 = dst2src.ptr<float>(1)[0];
    m_dst2src.v4 = dst2src.ptr<float>(1)[1];
    m_dst2src.v5 = dst2src.ptr<float>(1)[2];

    m_src2dst.v0 = src2dst.ptr<float>(0)[0];
    m_src2dst.v1 = src2dst.ptr<float>(0)[1];
    m_src2dst.v2 = src2dst.ptr<float>(0)[2];
    m_src2dst.v3 = src2dst.ptr<float>(1)[0];
    m_src2dst.v4 = src2dst.ptr<float>(1)[1];
    m_src2dst.v5 = src2dst.ptr<float>(1)[2];

    return true;
}

void u2net::U2NET::check()
{
    int idx;
    nvinfer1::Dims dims;

    sample::gLogInfo << "the engine's info:" << std::endl;
    for (auto layer_name : m_param.input_output_names)
    {
        idx = this->m_engine->getBindingIndex(layer_name.c_str());
        dims = this->m_engine->getBindingDimensions(idx);
        sample::gLogInfo << "idx = " << idx << ", " << layer_name << ": ";
        for (int i = 0; i < dims.nbDims; i++)
        {
            sample::gLogInfo << dims.d[i] << ", ";
        }
        sample::gLogInfo << std::endl;
    }

    sample::gLogInfo << "the context's info:" << std::endl;
    for (auto layer_name : m_param.input_output_names)
    {
        idx = this->m_engine->getBindingIndex(layer_name.c_str());
        dims = this->m_context->getBindingDimensions(idx);
        sample::gLogInfo << "idx = " << idx << ", " << layer_name << ": ";
        for (int i = 0; i < dims.nbDims; i++)
        {
            sample::gLogInfo << dims.d[i] << ", ";
        }
        sample::gLogInfo << std::endl;
    }
}

void u2net::U2NET::copy(const std::vector<cv::Mat>& imgsBatch)
{
    cv::Mat img_fp32 = cv::Mat::zeros(imgsBatch[0].size(), CV_32FC3); // todo 
    hipHostRegister(img_fp32.data, img_fp32.elemSize() * img_fp32.total(), hipHostRegisterPortable);

    // copy to device
    float* pi = m_input_src_device;
    //for (size_t i = 0; i < m_param.batch_size; i++)
    for (size_t i = 0; i < imgsBatch.size(); i++)
    {
        //std::vector<float> img_vec = std::vector<float>(imgsBatch[i].reshape(1, 1));
        imgsBatch[i].convertTo(img_fp32, CV_32FC3);
        CHECK(hipMemcpy(pi, img_fp32.data, sizeof(float) * 3 * m_param.src_h * m_param.src_w, hipMemcpyHostToDevice));
        /*imgsBatch[i].convertTo(imgsBatch[i], CV_32FC3);
        CHECK(hipMemcpy(pi, imgsBatch[i].data, sizeof(float) * 3 * m_param.src_h * m_param.src_w, hipMemcpyHostToDevice));*/
        pi += 3 * m_param.src_h * m_param.src_w;
    }

    hipHostUnregister(img_fp32.data);
}

void u2net::U2NET::preprocess(const std::vector<cv::Mat>& imgsBatch)
{
    bgr2rgbDevice(m_param.batch_size, m_input_src_device, m_param.src_w, m_param.src_h,
        m_input_rgb_device, m_param.src_w, m_param.src_h);
    resizeDevice(m_param.batch_size, m_input_rgb_device, m_param.src_w, m_param.src_h,
        m_input_resize_device, m_param.dst_w, m_param.dst_h, utils::ColorMode::RGB, m_dst2src);
    float* p_tmp = m_input_resize_device;
    float* p_max = m_max_val_device;
    for (size_t i = 0; i < imgsBatch.size(); i++)
    {
        float* max_dev = thrust::max_element(thrust::device, p_tmp, p_tmp + m_param.dst_h * m_param.dst_w);
        p_tmp += m_param.dst_h * m_param.dst_w;
        CHECK(hipMemcpy(p_max++, max_dev, sizeof(float), hipMemcpyDeviceToDevice));

    }
    u2netDivMaxDevice(m_param.batch_size, m_input_resize_device, m_param.dst_w, m_param.dst_h, 3, m_max_val_device);
    normDevice(m_param.batch_size, m_input_resize_device, m_param.dst_w, m_param.dst_h,
        m_input_norm_device, m_param.dst_w, m_param.dst_h, m_param);
    hwc2chwDevice(m_param.batch_size, m_input_norm_device, m_param.dst_w, m_param.dst_h,
        m_input_hwc_device, m_param.dst_w, m_param.dst_h);
}

bool u2net::U2NET::infer()
{
    float* bindings[] = { m_input_hwc_device, m_output_src_device };
    bool context = m_context->executeV2((void**)bindings);
    return context;
}

void u2net::U2NET::postprocess(const std::vector<cv::Mat>& imgsBatch)
{
    float* p_tmp = m_output_src_device;
    float* p_max = m_max_val_device;
    float* p_min = m_min_val_device;
    for (size_t i = 0; i < imgsBatch.size(); i++)
    {
        thrust::pair<float*, float*> min_max_dev = thrust::minmax_element(thrust::device, p_tmp, p_tmp + m_param.dst_h * m_param.dst_w);
        p_tmp += m_param.dst_h * m_param.dst_w;
        CHECK(hipMemcpy(p_min++, min_max_dev.first, sizeof(float), hipMemcpyDeviceToDevice));
        CHECK(hipMemcpy(p_max++, min_max_dev.second, sizeof(float), hipMemcpyDeviceToDevice));
    }
    u2netNormPredDevice(m_param.batch_size, m_output_src_device, m_param.dst_w, m_param.dst_h, 255.f, m_min_val_device, m_max_val_device);
    resizeDevice(m_param.batch_size, m_output_src_device, m_param.dst_w, m_param.dst_h,
        m_output_resize_device, m_param.src_w, m_param.src_h, utils::ColorMode::GRAY, m_src2dst);
}

void u2net::U2NET::showMask(const std::vector<cv::Mat>& imgsBatch, const int& cvDelayTime)
{
    float* output_mask_device = m_output_resize_device;
    for (size_t j = 0; j < imgsBatch.size(); j++)
    {
        CHECK(hipMemcpy(m_output_mask_host, output_mask_device + j * m_param.src_w * m_param.src_h,
            sizeof(float) * m_param.src_w * m_param.src_h, hipMemcpyDeviceToHost));
        cv::Mat img_mask(m_param.src_h, m_param.src_w, CV_32FC1, m_output_mask_host);
        img_mask.convertTo(img_mask, CV_8UC1);
        cv::imshow("img_mask", img_mask);
        cv::waitKey(cvDelayTime);
    }
}

void u2net::U2NET::saveMask(const std::vector<cv::Mat>& imgsBatch, const std::string& savePath, const int& batchSize, const int& batchi)
{
    float* output_mask_device = m_output_resize_device;
    for (size_t j = 0; j < imgsBatch.size(); j++)
    {
        CHECK(hipMemcpy(m_output_mask_host, output_mask_device + j * m_param.src_w * m_param.src_h,
            sizeof(float) * m_param.src_w * m_param.src_h, hipMemcpyDeviceToHost));
        cv::Mat img_mask(m_param.src_h, m_param.src_w, CV_32FC1, m_output_mask_host);
        img_mask.convertTo(img_mask, CV_8UC1);
        int imgi = batchi * batchSize + j;
		cv::imwrite(savePath + "_" + std::to_string(imgi) + ".jpg", img_mask);
		cv::waitKey(1);
    }
}

void u2net::U2NET::reset()
{
}

__global__
void u2net_div_max_device_kernel(int batch_size, float* src, int img_height, int img_width, int img_volume, float* maxVals)
{
    int dx = blockDim.x * blockIdx.x + threadIdx.x;
    int dy = blockDim.y * blockIdx.y + threadIdx.y;
    if (dx < img_volume && dy < batch_size)
    {
        src[dy * img_volume + dx] /= maxVals[dy];
    }
}

static __device__
float u2net_norm_device(float val, float scale, float min_val, float max_val)
{
    return scale * (val  - min_val) / (max_val - min_val);
}

__global__
void u2net_norm_pred_device_kernel(int batch_size, float* src, int img_height, int img_width, int img_area, float scale, float* minVals, float* maxVals)
{
    int dx = blockDim.x * blockIdx.x + threadIdx.x;
    int dy = blockDim.y * blockIdx.y + threadIdx.y;
    if (dx < img_area && dy < batch_size)
    {
        src[dy * img_area + dx] = u2net_norm_device(src[dy * img_area + dx], scale, minVals[dy], maxVals[dy]);
    }
}

void u2netDivMaxDevice(const int& batchSize, float* src, int srcWidth, int srcHeight, int channel, float* maxVals)
{
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_size((srcWidth * srcHeight * channel + BLOCK_SIZE - 1) / BLOCK_SIZE,
        (batchSize + BLOCK_SIZE - 1) / BLOCK_SIZE);

    int img_volume = channel * srcHeight * srcWidth;
    int img_height = srcHeight;
    int img_width = srcWidth;
    u2net_div_max_device_kernel << < grid_size, block_size, 0, nullptr >> > (batchSize, src, img_height, img_width, img_volume, maxVals);
}

void u2netNormPredDevice(const int& batchSize, float* src, int srcWidth, int srcHeight, float scale, float* minVals, float* maxVals)
{
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_size((srcWidth * srcHeight * 1 + BLOCK_SIZE - 1) / BLOCK_SIZE,
        (batchSize + BLOCK_SIZE - 1) / BLOCK_SIZE);

    int img_area = srcHeight * srcWidth;
    int img_height = srcHeight;
    int img_width = srcWidth;
    u2net_norm_pred_device_kernel << < grid_size, block_size, 0, nullptr >> > (batchSize, src, img_height, img_width, img_area, scale, minVals, maxVals);
}